#include "hip/hip_runtime.h"
//3090
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
//#include <conio.h>
#include <time.h>
using namespace std;

#define M 128
#define N 64
#define MaxThreadPerBlock 1024
#define Blocknum 4096//1time:2^22  18min:2^38  1day:2^44-2^45
int array1[64];
int results[64] = {0}; //保存已经找到的结果前缀数组
int results_end = 0; //结果前缀数组有效数据下标
int64_t res=0;
int64_t num=0;
int64_t simutime=0;
clock_t start;
clock_t endtime;
int64_t* d_A, * d_B,* h_B;
int* d_C, * d_resultnum;
int* resultnum,* d_weight,*weight;
/**
 * 组合
 * @param deep 递归深度
 * @param n 数组最大长度
 * @param m 要查找的组合的长度
 */
 FILE* fp_16;
 FILE* fp_18;
 FILE* fp_20;
 FILE* fp_22;
 FILE* fp_test;

 void checkrepeat(FILE *f,int name,int mode)
{
    uint64_t num;
    int k=0;
    int sum=0;
    uint64_t results[1000000];
    fseek(f,0,0);
    for(uint64_t i=0;!feof(f);i++)
    {
        k=0;
        fscanf(f, "%ld", &num);
        // for(uint64_t t=0;t<sum;t++)
        // {
        //     if(num==results[t])
        //     {
        //         k=1;
        //             //cout<<num<<endl;
        //         break;
        //     }
        // }
        if(k==0)
        {
            results[sum]=num;
            sum=sum+1;
            //cout<<sum<<endl;
        }
    }
    fclose(f);
    if(mode==1)
    {
        if(name==16)
        {
            f=fopen("16_1.txt","w+");
        }
        else if(name==18)
        {
            f=fopen("18_1.txt","w+");
        }
        else if(name==20)
        {
            f=fopen("20_1.txt","w+");
        }
        else if(name==22)
        {
            f=fopen("22_1.txt","w+");
        }
        for(uint64_t i=0;i<sum;i++)
        {
            fprintf(f,"%ld ",results[i]);
        }
        fprintf(f,"\n\n");
        fclose(f);
        printf("saved\n");
    }
    if(name==16)
    {
        f=fopen("16.txt","a+");
    }
    else if(name==18)
    {
        f=fopen("18.txt","a+");
    }
    else if(name==20)
    {
        f=fopen("20.txt","a+");
    }
    else if(name==22)
    {
        f=fopen("22.txt","a+");
    }
    // for(uint64_t i=0;i<sum;i++)
    // {
    //     fprintf(f,"%ld ",results[i]);
    // }
    // fprintf(f,"\n\n");
    //fclose(f);
    //return sum;
}

__device__ int64_t hamming(int64_t n) {
    n = (n & 0x5555555555555555) + ((n >> 1) & 0x5555555555555555);
    n = (n & 0x3333333333333333) + ((n >> 2) & 0x3333333333333333);
    n = (n & 0x0f0f0f0f0f0f0f0f) + ((n >> 4) & 0x0f0f0f0f0f0f0f0f);
    n = (n & 0x00ff00ff00ff00ff) + ((n >> 8) & 0x00ff00ff00ff00ff);
    n = (n & 0x0000ffff0000ffff) + ((n >> 16) & 0x0000ffff0000ffff);
    n = (n & 0x00000000ffffffff) + ((n >> 32) & 0x00000000ffffffff);
    return n;
}

__global__ void simu(int64_t* a, int64_t* b, int* c, int t,int* weight)//���ж������
{
    int offset;
    offset = threadIdx.x + blockIdx.x * blockDim.x;  
    __shared__ int k;
    k= offset / 128 + t * MaxThreadPerBlock * Blocknum / 128;   
    //c[k * (M + 1) + offset%128] = hamming(a[offset%128] & b[k])%2;  
    atomicAdd(&weight[k], hamming(a[offset%128] & b[k])%2);    
}
__global__ void countweight(int64_t* b,int* c,int* result1,int64_t num)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    if(k<num)
    {
         if (c[k] < 23)
         {
            atomicAdd(&result1[c[k]],1);
         }
    }
}
__global__ void clearzero(int* c)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    c[k] = 0;
}
void comb(int deep, int n, int m,int64_t* b)
 {
     if (deep > n) // 越界递归结束
         return;
     if (results_end == m) // 找到结果，打印，递归结束
     {
         res=0;
         for (int i = 0; i < m; i++)
         {
             
             res=res|((int64_t)1<<results[i]);
             
         }
         b[num]=res;
         num++;
         if(num==MaxThreadPerBlock*Blocknum)
         {
            hipMemcpy(d_B, b, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyHostToDevice);
            for (int t = 0; t < 128; t++)
            {
                simu << <Blocknum * MaxThreadPerBlock / 128, 128 >> > (d_A, d_B, d_C, t,d_weight);
            }
            countweight << < Blocknum, MaxThreadPerBlock >> > (d_B,d_weight,d_resultnum,num);
            num=0;
            simutime++;
            hipMemcpy(weight, d_weight, Blocknum * MaxThreadPerBlock * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyDeviceToHost);
            for(int i=0;i < MaxThreadPerBlock * Blocknum;i++)
            {
                
                    if(weight[i]==16)
                    {
                        fprintf(fp_16,"%ld ",h_B[i]);
                    }
                    else if(weight[i]==18)
                    {
                        fprintf(fp_18,"%ld ",h_B[i]);
                    }
                    else if(weight[i]==20)
                    {
                        fprintf(fp_20,"%ld ",h_B[i]);
                    }
                    else if(weight[i]==22)
                    {
                        fprintf(fp_22,"%ld ",h_B[i]);
                    }
            }
            clearzero << < Blocknum, MaxThreadPerBlock >> > (d_weight);
            if(simutime%1024==0)
            {
                if(simutime%4096==0)
                {
                    checkrepeat(fp_16,16,1);
                    checkrepeat(fp_18,18,1);
                    checkrepeat(fp_20,20,1);
                    checkrepeat(fp_22,22,1);
                }
                endtime = clock();
                printf("\ntime=%f min\n", (double)(endtime - start) / CLOCKS_PER_SEC/60);
           
                hipMemcpy(resultnum, d_resultnum, 23 * sizeof(int), hipMemcpyDeviceToHost);
                fprintf(fp_test,"simutimes = %ld\n",simutime);
                for (int i = 0; i < 23; i++)
                {
                    cout << resultnum[i] << "  ";
                    fprintf(fp_test,"%d ",resultnum[i]);
                }
                cout << endl;
                cout<<"i = "<<m<<endl;
                cout << simutime << endl;
            }
         }
         return;
     }
     results[results_end++] = array1[deep];
     comb(deep+1, n, m,b); //向下一级递归
     results_end--;
     comb(deep+1, n, m,b); //向下一级递归
 }
int main()
{
    hipError_t cudaStatus;
    hipDeviceProp_t prop;
    int Num_Device;

    // ����ϵͳ�е�GPU����,��ָ��������һ��,ͬʱ�õ���GPU�����ܲ���
    cudaStatus = hipGetDeviceCount(&Num_Device);
    if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
    {
        printf("There is no GPU beyond 1.0, exit!\n");
        exit(0);
    }
    else
    {
        cudaStatus = hipGetDeviceProperties(&prop, Num_Device - 1);	// ѡ�����һ��GPU���ڼ���,ͬʱ����������ܲ���
        if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
        {
            printf("Cannot get device properties, exit!\n");
            exit(0);
        }
    }
    printf("Device Name : %s.\n", prop.name);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
    printf("maxThreadsPerMultiProcessor : %d.\n", prop.maxThreadsPerMultiProcessor);
    printf("Blocknum : %d.\n", Blocknum);
    int* h_AT = (int*)malloc(sizeof(int) * M * N);
    int64_t* h_A = (int64_t*)malloc(sizeof(int64_t) * M);
    h_B = (int64_t*)malloc(sizeof(int64_t) * Blocknum * MaxThreadPerBlock);
    int* h_C = (int*)malloc(sizeof(int) * Blocknum * MaxThreadPerBlock * M);
    weight = (int*)malloc(sizeof(int) * Blocknum * MaxThreadPerBlock);
    resultnum = (int*)malloc(sizeof(int) * 23);
    for (int i = 0; i < 23; i++)
    {
        resultnum[i] = 0;
    }

    fp_test = fopen("G_SPP.txt", "r");
    int h_num = 0;
    for (int i = 0; i < M * N; i++)
    {

        fscanf(fp_test, "%d", &h_num);//ÿM��Ԫ�أ�ֻ��¼һ����1��λ��        
        h_AT[i] = h_num;
    }
    fclose(fp_test);
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_A[i] = 2 * h_A[i] + h_AT[(63-j)*M + i];
        }
    }

    for(int k=0;k<64;k++)
    {
        array1[k]=k;
    }
    hipMalloc((void**)&d_A, sizeof(int64_t) * M);
    hipMalloc((void**)&d_B, sizeof(int64_t) * Blocknum * MaxThreadPerBlock);
    hipMalloc((void**)&d_C, sizeof(int) * Blocknum * MaxThreadPerBlock * M);
    hipMalloc((void**)&d_resultnum, sizeof(int) * 23);
    hipMalloc((void**)&d_weight, sizeof(int) * Blocknum * MaxThreadPerBlock);

    hipMemcpy(d_A, h_A, M * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_resultnum, resultnum, 23 * sizeof(int), hipMemcpyHostToDevice);
    start = clock();
    fp_16=fopen("16.txt","w+");
    fp_18=fopen("18.txt","w+");
    fp_20=fopen("20.txt","w+");
    fp_22=fopen("22.txt","w+");
    fp_test=fopen("results.txt","w");
    fprintf(fp_16,"\n\n");
    fprintf(fp_18,"\n\n");
    fprintf(fp_20,"\n\n");
    fprintf(fp_22,"\n\n");
    for(int i=1; i<=13; i++){
        comb(0, 64, i, h_B);
        cout<<"i= "<<i<<endl<<endl;
    }    
    if(num!=0)
    {
        hipMemcpy(d_B, h_B, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyHostToDevice);
        for (int t = 0; t < 128; t++)
        {
            simu << <Blocknum * MaxThreadPerBlock / 128, 128 >> > (d_A, d_B, d_C, t,d_weight);
        }
        countweight << < Blocknum, MaxThreadPerBlock >> > (d_B,d_weight,d_resultnum,num);
        hipMemcpy(weight, d_weight, Blocknum * MaxThreadPerBlock * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyDeviceToHost);
        for(int i=0;i < MaxThreadPerBlock * Blocknum;i++)
        {
            
                if(weight[i]==16)
                {
                    fprintf(fp_16,"%ld ",h_B[i]);
                }
                else if(weight[i]==18)
                {
                    fprintf(fp_18,"%ld ",h_B[i]);
                }
                else if(weight[i]==20)
                {
                    fprintf(fp_20,"%ld ",h_B[i]);
                }
                else if(weight[i]==22)
                {
                    fprintf(fp_22,"%ld ",h_B[i]);
                }
        }
        clearzero << < Blocknum, MaxThreadPerBlock >> > (d_weight);

    }
    endtime = clock();
    printf("\ntime=%f min", (double)(endtime - start) / CLOCKS_PER_SEC/60);
    printf("\nsimutime=%ld times", simutime);
    printf("\nextranum=%ld times", num); 
    hipMemcpy(resultnum, d_resultnum, 23 * sizeof(int), hipMemcpyDeviceToHost);
    cout<<endl;
    fprintf(fp_test,"simutime = %ld\n",simutime);
    fprintf(fp_test,"extranum = %ld\n",num);
    for (int i = 0; i < 23; i++)
    {
        cout << resultnum[i] << "  ";
        fprintf(fp_test,"%d ",resultnum[i]);
    }
    fprintf(fp_test,"\n");
    fclose(fp_16);
    fclose(fp_18);
    fclose(fp_20);
    fclose(fp_22);
    fclose(fp_test);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_AT);
    free(h_B);
    free(h_C);
    return 0;
}


