#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
//#include <conio.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#define M 128
#define N 64
#define MaxThreadPerBlock 1024
#define Blocknum 4096//1time:2^22  18min:2^38  1day:2^44-2^45
int64_t checkrepeat(FILE *f,int name,int mode)
{
    int64_t num;
    int k=0;
    int sum=0;
    int64_t results[1000000];
    fseek(f,0,0);
    for(int64_t i=0;!feof(f);i++)
    {
        k=0;
        fscanf(f, "%ld", &num);
        for(int64_t t=0;t<sum;t++)
        {
            if(num==results[t])
            {
                k=1;
                    //cout<<num<<endl;
                break;
            }
        }
        if(k==0)
        {
            results[sum]=num;
            sum=sum+1;
            //cout<<sum<<endl;
        }
    }
    fclose(f);
    if(mode==1)
    {
        if(name==16)
        {
            f=fopen("16_1.txt","w+");
        }
        else if(name==18)
        {
            f=fopen("18_1.txt","w+");
        }
        else if(name==20)
        {
            f=fopen("20_1.txt","w+");
        }
        else if(name==22)
        {
            f=fopen("22_1.txt","w+");
        }
        for(int64_t i=0;i<sum;i++)
        {
            fprintf(f,"%ld ",results[i]);
        }
        fprintf(f,"\n\n");
        fclose(f);
        printf("saved\n");
    }
    if(name==16)
    {
        f=fopen("16.txt","w+");
    }
    else if(name==18)
    {
        f=fopen("18.txt","w+");
    }
    else if(name==20)
    {
        f=fopen("20.txt","w+");
    }
    else if(name==22)
    {
        f=fopen("22.txt","w+");
    }
    for(int64_t i=0;i<sum;i++)
    {
        fprintf(f,"%ld ",results[i]);
    }
    fprintf(f,"\n\n");
    //fclose(f);
    return sum;
}
int64_t reverse(int64_t a)
{
    int64_t b=0;
    int64_t temp=1;
    for(int i=0;i<64;i++)
    {
        b=b*2+((a>>i)&temp);
    }
    return b;
}
__device__ int64_t hamming(int64_t n) {
    n = (n & 0x5555555555555555) + ((n >> 1) & 0x5555555555555555);
    n = (n & 0x3333333333333333) + ((n >> 2) & 0x3333333333333333);
    n = (n & 0x0f0f0f0f0f0f0f0f) + ((n >> 4) & 0x0f0f0f0f0f0f0f0f);
    n = (n & 0x00ff00ff00ff00ff) + ((n >> 8) & 0x00ff00ff00ff00ff);
    n = (n & 0x0000ffff0000ffff) + ((n >> 16) & 0x0000ffff0000ffff);
    n = (n & 0x00000000ffffffff) + ((n >> 32) & 0x00000000ffffffff);
    return n;
}

__global__ void simu(int64_t* a, int64_t* b, int* c,int t,int* weight)//���ж������
{
    int offset;
    offset = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ int k;
    k= offset / 128 + t * MaxThreadPerBlock * Blocknum / 128;
    //c[k * M + offset%128] = hamming(a[offset%128] & b[k])%2;  
    atomicAdd(&weight[k], hamming(a[offset%128] & b[k])%2);
    __syncthreads();
}
__global__ void hbadd(int64_t* b,int64_t rand)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    b[k] = 0;
    int r = 0;
    int r1=0;
    int t=0;
    int64_t weight=0;
    hiprandState state;
    hiprand_init((int64_t)rand*4194304+k,0,0,&state);
    r=hiprand(&state)%10+1;
    for(int i=0;i<r;i++)
    {
       while(weight==hamming(b[k])&&t<10)
        {
            t++;
            r1=hiprand(&state)%64;
            b[k]=b[k]|((int64_t)1<<r1);
        }
        t=0;
        weight=hamming(b[k]);  
    }
    __syncthreads();

}
__global__ void clearzero(int* c)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    c[k] = 0;
    __syncthreads();
}
int main()
{
    srand((unsigned int)time(NULL));
    hipError_t cudaStatus;
    hipDeviceProp_t prop;
    FILE* fp_test;
    int Num_Device;

    // ����ϵͳ�е�GPU����,��ָ��������һ��,ͬʱ�õ���GPU�����ܲ���
    cudaStatus = hipGetDeviceCount(&Num_Device);
    if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
    {
        printf("There is no GPU beyond 1.0, exit!\n");
        exit(0);
    }
    else
    {
        cudaStatus = hipGetDeviceProperties(&prop, Num_Device - 1);	// ѡ�����һ��GPU���ڼ���,ͬʱ����������ܲ���
        if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
        {
            printf("Cannot get device properties, exit!\n");
            exit(0);
        }
    }
    printf("Device Name : %s.\n", prop.name);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
    printf("maxThreadsPerMultiProcessor : %d.\n", prop.maxThreadsPerMultiProcessor);
    printf("Blocknum : %d.\n", Blocknum);
    int* h_AT = (int*)malloc(sizeof(int) * M * N);
    int64_t* h_A = (int64_t*)malloc(sizeof(int64_t) * M);
    int64_t* h_A1 = (int64_t*)malloc(sizeof(int64_t) * M);
    int64_t* h_B = (int64_t*)malloc(sizeof(int64_t) * Blocknum * MaxThreadPerBlock);
    int* h_C = (int*)malloc(sizeof(int) * Blocknum * MaxThreadPerBlock * M);
    int* weight = (int*)malloc(sizeof(int) * Blocknum * MaxThreadPerBlock);
    int* resultnum = (int*)malloc(sizeof(int) * 23);
   
    for (int i = 0; i < 23; i++)
    {
        resultnum[i] = 0;
    }

    fp_test = fopen("G_PAC.txt", "r");
    int h_num = 0;
    for (int i = 0; i < M * N; i++)
    {

        fscanf(fp_test, "%d", &h_num);//ÿM��Ԫ�أ�ֻ��¼һ����1��λ��        
        h_AT[i] = h_num;
    }
    fclose(fp_test);
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_A[i] = 2 * h_A[i] + h_AT[(63-j) * M + i];
            //h_A1[i] = 2 * h_A1[i] + h_AT[j* M + i];
        }
    }
    // for (int i = 0; i < Blocknum * MaxThreadPerBlock; i++)
    // {
    //     h_B[i] = i+1;   
    // }

    int64_t* d_A, * d_B;
    int* d_C,* d_weight;
    hipMalloc((void**)&d_weight, sizeof(int) * Blocknum * MaxThreadPerBlock);
    hipMalloc((void**)&d_A, sizeof(int64_t) * M);
    //hipMalloc((void**)&d_A1, sizeof(int64_t) * M);
    hipMalloc((void**)&d_B, sizeof(int64_t) * Blocknum * MaxThreadPerBlock);
    hipMalloc((void**)&d_C, sizeof(int) * Blocknum * MaxThreadPerBlock * M);
    //hipMalloc((void**)&d_resultnum, sizeof(int) * 23);
   


    hipMemcpy(d_A, h_A, M * sizeof(int64_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_A1, h_A1, M * sizeof(int64_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_B, h_B, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyHostToDevice);
    //hipMemcpy(d_resultnum, resultnum, 23 * sizeof(int), hipMemcpyHostToDevice);
    int64_t k = 0;
    clock_t start = clock();
    clock_t end;
    FILE* fp_16;
    FILE* fp_18;
    FILE* fp_20;
    FILE* fp_22;
    fp_16=fopen("16.txt","a+");
    fp_18=fopen("18.txt","a+");
    fp_20=fopen("20.txt","a+");
    fp_22=fopen("22.txt","a+");
    fp_test=fopen("results.txt","a");
    fprintf(fp_16,"\n\n");
    fprintf(fp_18,"\n\n");
    fprintf(fp_20,"\n\n");
    fprintf(fp_22,"\n\n");
    while (k<1024)//��
    //while(k<256)
    {
        //cout<<(unsigned int)time(NULL)<<endl<<endl;
        hbadd << <Blocknum * MaxThreadPerBlock / 128, 128 >> > (d_B,k+(unsigned int)time(NULL));
        for (int i = 0; i < 128; i++)
        {
            simu << <Blocknum * MaxThreadPerBlock / 128, 128 >> > (d_A, d_B,d_C,i,d_weight);
        }
       
        //hipDeviceSynchronize();
        //countweight << < Blocknum, MaxThreadPerBlock >> > (d_weight,d_resultnum);
//       hipMemcpy(h_C, d_C, Blocknum * MaxThreadPerBlock * (M + 1) * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(weight, d_weight, Blocknum * MaxThreadPerBlock * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();        
        for(int i=0;i < MaxThreadPerBlock * Blocknum;i++)
        {
            
                if(weight[i]==16)
                {
                    fprintf(fp_16,"%ld ",h_B[i]);
                }
                else if(weight[i]==18)
                {
                    fprintf(fp_18,"%ld ",h_B[i]);
                }
                else if(weight[i]==20)
                {
                    fprintf(fp_20,"%ld ",h_B[i]);
                }
                else if(weight[i]==22)
                {
                    fprintf(fp_22,"%ld ",h_B[i]);
                }
        }
        clearzero << < Blocknum, MaxThreadPerBlock >> > (d_weight);
        if ((k+1)%256==0)
        {
            //srand((unsigned int)time(NULL));
            if ((k+1)%1024)
            {
                resultnum[16]=checkrepeat(fp_16,16,0);
                resultnum[18]=checkrepeat(fp_18,18,0);
                resultnum[20]=checkrepeat(fp_20,20,0);
                resultnum[22]=checkrepeat(fp_22,22,0);
            }
            else
            {
                resultnum[16]=checkrepeat(fp_16,16,1);
                resultnum[18]=checkrepeat(fp_18,18,1);
                resultnum[20]=checkrepeat(fp_20,20,1);
                resultnum[22]=checkrepeat(fp_22,22,1);
            }
            end = clock();
            printf("\ntime=%f min\n", (double)(end - start) / CLOCKS_PER_SEC/60);
           
            //hipMemcpy(resultnum, d_resultnum, 23 * sizeof(int), hipMemcpyDeviceToHost);
            fprintf(fp_test,"k = %ld\n",k);
            for (int i = 0; i < 23; i++)
            {
                cout << resultnum[i] << "  ";
                fprintf(fp_test,"%d ",resultnum[i]);
            }
            cout << endl;
            fprintf(fp_test,"\n");
            cout << k << endl;           
            fclose(fp_test);
            fp_test=fopen("results.txt","a");
        }
        k++;
        //cout<<k<<endl;

    }
    resultnum[16]=checkrepeat(fp_16,16,1);
    resultnum[18]=checkrepeat(fp_18,18,1);
    resultnum[20]=checkrepeat(fp_20,20,1);
    resultnum[22]=checkrepeat(fp_22,22,1);
    end = clock();
    printf("\ntime=%f min\n", (double)(end - start) / CLOCKS_PER_SEC/60);
    hipMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, Blocknum * MaxThreadPerBlock * M * sizeof(int), hipMemcpyDeviceToHost);
    cout << h_B[Blocknum * MaxThreadPerBlock - 1] << endl;
    //hipMemcpy(resultnum, d_resultnum, 23 * sizeof(int), hipMemcpyDeviceToHost);
    cout<<endl;
    // for(int i=0;i<1024;i++)
    // {
    //     //fprintf(fp_test,"%ld ",h_B[i]);
    //     cout<<h_B[i]<<endl;
    // }
    // cout<<endl;
    // for(int i=0;i<2000;i++)
    // {
    //     cout<<h_C[i]<<" ";
    //     if((i+1)%128==0)
    //     {
    //         cout<<endl<<endl;
    //     }
    // }
    fprintf(fp_test,"k = %ld\n",k);
    for (int i = 0; i < 23; i++)
    {
        cout << resultnum[i] << "  ";
        fprintf(fp_test,"%d ",resultnum[i]);
    }
    fprintf(fp_test,"\n");
    cout << endl;
    fclose(fp_16);
    fclose(fp_18);
    fclose(fp_20);
    fclose(fp_22);
    fclose(fp_test);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_AT);
    free(h_B);
    free(h_C);
    return 0;
}