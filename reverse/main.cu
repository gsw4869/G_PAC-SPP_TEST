#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
//#include <conio.h>
#include <time.h>
using namespace std;

#define M 224
#define N 64
#define MaxThreadPerBlock 1024
#define Blocknum 224 //1time:2^22  18min:2^38  1day:2^44-2^45

void checkrepeat(FILE *f, int name, int mode)
{
    uint64_t num;
    int k = 0;
    int sum = 0;
    uint64_t results[1000000];
    fseek(f, 0, 0);
    for (uint64_t i = 0; !feof(f); i++)
    {
        k = 0;
        fscanf(f, "%ld", &num);
        if (k == 0)
        {
            results[sum] = num;
            sum = sum + 1;
            //cout<<sum<<endl;
        }
    }
    fclose(f);
    if (mode == 1)
    {
        if (name == 16)
        {
            f = fopen("16_1.txt", "w+");
        }
        else if (name == 18)
        {
            f = fopen("18_1.txt", "w+");
        }
        else if (name == 20)
        {
            f = fopen("20_1.txt", "w+");
        }
        else if (name == 22)
        {
            f = fopen("22_1.txt", "w+");
        }
        for (uint64_t i = 0; i < sum; i++)
        {
            fprintf(f, "%ld ", results[i]);
        }
        fprintf(f, "\n\n");
        fclose(f);
        printf("saved\n");
    }
    if (name == 16)
    {
        f = fopen("16.txt", "a+");
    }
    else if (name == 18)
    {
        f = fopen("18.txt", "a+");
    }
    else if (name == 20)
    {
        f = fopen("20.txt", "a+");
    }
    else if (name == 22)
    {
        f = fopen("22.txt", "a+");
    }
}

__device__ int64_t hamming(uint64_t n)
{
    n = (n & 0x5555555555555555) + ((n >> 1) & 0x5555555555555555);
    n = (n & 0x3333333333333333) + ((n >> 2) & 0x3333333333333333);
    n = (n & 0x0f0f0f0f0f0f0f0f) + ((n >> 4) & 0x0f0f0f0f0f0f0f0f);
    n = (n & 0x00ff00ff00ff00ff) + ((n >> 8) & 0x00ff00ff00ff00ff);
    n = (n & 0x0000ffff0000ffff) + ((n >> 16) & 0x0000ffff0000ffff);
    n = (n & 0x00000000ffffffff) + ((n >> 32) & 0x00000000ffffffff);
    return n;
}

__global__ void simu(uint64_t *a, uint64_t *b, int *c, int t, int *weight) //���ж������?
{
    int offset;
    offset = threadIdx.x + blockIdx.x * blockDim.x;
    if (offset < MaxThreadPerBlock * Blocknum)
    {
        int k;
        k = offset / M + t * MaxThreadPerBlock * Blocknum / M;
        if (b[k] < pow(2,63))
        {
            //c[k * M + offset%128] = hamming(a[offset%128] & b[k])%2;
            atomicAdd(&weight[k], hamming(a[offset % M] & b[k]) % 2);
        }
    }
}
__global__ void countweight(int *c, int *result)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    // if (c[k] < 23)
    // {
    atomicAdd(&result[c[k]], 1);
    // }
}
__global__ void hbadd(uint64_t *b)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    if (k < MaxThreadPerBlock * Blocknum)
    {
        b[k] += MaxThreadPerBlock * Blocknum;
    }
}
__global__ void clearzero(int *c)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    c[k] = 0;
}
int main()
{
    hipError_t cudaStatus;
    hipDeviceProp_t prop;
    FILE *fp_test;
    int Num_Device;

    // ����ϵͳ�е�GPU����,��ָ��������һ��,ͬʱ�õ���GPU�����ܲ���
    cudaStatus = hipGetDeviceCount(&Num_Device);
    if (cudaStatus != hipSuccess) // û��һ��������ڼ����GPU,���������в����޷�����
    {
        printf("There is no GPU beyond 1.0, exit!\n");
        exit(0);
    }
    else
    {
        cudaStatus = hipGetDeviceProperties(&prop, Num_Device - 1); // ѡ�����һ��GPU���ڼ���,ͬʱ����������ܲ���?
        if (cudaStatus != hipSuccess)                               // û��һ��������ڼ����GPU,���������в����޷�����
        {
            printf("Cannot get device properties, exit!\n");
            exit(0);
        }
    }
    printf("Device Name : %s.\n", prop.name);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
    printf("maxThreadsPerMultiProcessor : %d.\n", prop.maxThreadsPerMultiProcessor);
    printf("Blocknum : %d.\n", Blocknum);
    int *h_AT = (int *)malloc(sizeof(int) * M * N);
    uint64_t *h_A = (uint64_t *)malloc(sizeof(uint64_t) * M);
    uint64_t *h_B = (uint64_t *)malloc(sizeof(uint64_t) * Blocknum * MaxThreadPerBlock);
    int *h_C = (int *)malloc(sizeof(int) * Blocknum * MaxThreadPerBlock * M);
    int *weight = (int *)malloc(sizeof(int) * Blocknum * MaxThreadPerBlock);
    int *resultnum = (int *)malloc(sizeof(int) * 256);

    for (int i = 0; i < 256; i++)
    {
        resultnum[i] = 0;
    }

    fp_test = fopen("BG2_16_6_14_copy.txt", "r");
    int h_num = 0;
    for (int i = 0; i < M * N; i++)
    {
        fscanf(fp_test, "%d", &h_num); 
        h_AT[i] = h_num;
    }
    fclose(fp_test);

    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_A[i] = 2 * h_A[i] + h_AT[j * M + i];
        }
    }
    for (int i = 0; i < Blocknum * MaxThreadPerBlock; i++)
    {
        h_B[i] = i;
    }

    uint64_t *d_A, *d_B;
    int *d_C, *d_resultnum, *d_weight;
    hipMalloc((void **)&d_weight, sizeof(int) * Blocknum * MaxThreadPerBlock);
    hipMalloc((void **)&d_A, sizeof(uint64_t) * M);
    hipMalloc((void **)&d_B, sizeof(uint64_t) * Blocknum * MaxThreadPerBlock);
    hipMalloc((void **)&d_C, sizeof(int) * Blocknum * MaxThreadPerBlock * M);
    hipMalloc((void **)&d_resultnum, sizeof(int) * 256);

    hipMemcpy(d_A, h_A, M * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Blocknum * MaxThreadPerBlock * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_resultnum, resultnum, 256 * sizeof(int), hipMemcpyHostToDevice);
    uint64_t k = 0;
    clock_t start = clock();
    clock_t end;
    FILE *fp_16;
    FILE *fp_18;
    FILE *fp_20;
    FILE *fp_22;
    fp_16 = fopen("16.txt", "a+");
    fp_18 = fopen("18.txt", "a+");
    fp_20 = fopen("20.txt", "a+");
    fp_22 = fopen("22.txt", "a+");
    fp_test = fopen("results.txt", "a");
    int min_weight=255;
    while (k < pow(2,54)-1) //��
    {

        for (int i = 0; i < 224; i++)
        {
            simu<<<Blocknum * MaxThreadPerBlock / 224, 224>>>(d_A, d_B, d_C, i, d_weight);
        }
        hbadd<<<Blocknum * MaxThreadPerBlock / 224, 224>>>(d_B);

        //cudaThreadSynchronize();
        countweight<<<Blocknum, MaxThreadPerBlock>>>(d_weight, d_resultnum);
        //       cudaMemcpy(h_C, d_C, Blocknum * MaxThreadPerBlock * (M + 1) * sizeof(int), cudaMemcpyDeviceToHost);
        hipMemcpy(weight, d_weight, Blocknum * MaxThreadPerBlock * sizeof(int), hipMemcpyDeviceToHost);
        //       cudaMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(int), cudaMemcpyDeviceToHost);

#pragma omp parallel for
        for (int i = 0; i < MaxThreadPerBlock * Blocknum; i++)
        {
            if (weight[i] < min_weight && weight[i] != 0)
            {
                min_weight = weight[i];
                fp_16 = fopen("16.txt", "a+");
                fprintf(fp_16, "min_weight:%d, %ld \n", min_weight,h_B[i] + k * MaxThreadPerBlock * Blocknum);
                printf("min_weight:%d, %ld \n", min_weight,h_B[i] + k * MaxThreadPerBlock * Blocknum);
                fclose(fp_16);
            }
        }

        clearzero<<<Blocknum, MaxThreadPerBlock>>>(d_weight);
        if ((k + 1) % 1024 == 0)
        {
            // if ((k + 1) % 4096 == 0)
            // {
            //     checkrepeat(fp_16, 16, 1);
            //     checkrepeat(fp_18, 18, 1);
            //     checkrepeat(fp_20, 20, 1);
            //     checkrepeat(fp_22, 22, 1);
            // }
            end = clock();
            printf("\ntime=%f min\n", (double)(end - start) / CLOCKS_PER_SEC / 60);

            hipMemcpy(resultnum, d_resultnum, 256 * sizeof(int), hipMemcpyDeviceToHost);
            fprintf(fp_test, "k = %ld\n", k);
            for (int i = 0; i < 256; i++)
            {
                cout << "i =  " << i << " :";
                cout << resultnum[i] << "\n";
                fprintf(fp_test, "%d ", resultnum[i]);
            }
            cout << endl;
            fprintf(fp_test, "\n");
            cout << k << endl;
            fclose(fp_test);
            fp_test = fopen("results.txt", "a");
        }
        k++;
        //cudaThreadSynchronize();
    }
    end = clock();
    printf("\ntime=%f min\n", (double)(end - start) / CLOCKS_PER_SEC / 60);
    hipMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, Blocknum * MaxThreadPerBlock * M * sizeof(int), hipMemcpyDeviceToHost);
    cout << h_B[Blocknum * MaxThreadPerBlock - 1] << endl;
    hipMemcpy(resultnum, d_resultnum, 256 * sizeof(int), hipMemcpyDeviceToHost);
    cout << endl;
    // for(int i=0;i<10;i++)
    // {
    //     cout<<h_B[i]<<" ";
    // }
    // cout<<endl;
    // for(int i=0;i<2000;i++)
    // {
    //     cout<<h_C[i]<<" ";
    //     if((i+1)%128==0)
    //     {
    //         cout<<endl<<endl;
    //     }
    // }
    fprintf(fp_test, "k = %ld\n", k);
    for (int i = 0; i < 256; i++)
    {
        cout << "i =  " << i << " :";
        cout << resultnum[i] << "\n";
        fprintf(fp_test, "%d ", resultnum[i]);
    }
    fprintf(fp_test, "\n");
    cout << endl;
    fclose(fp_16);
    fclose(fp_18);
    fclose(fp_20);
    fclose(fp_22);
    fclose(fp_test);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_AT);
    free(h_B);
    free(h_C);
    return 0;
}