#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
//#include <conio.h>
#include <time.h>
using namespace std;

#define M 128
#define N 64
#define MaxThreadPerBlock 1024
#define Blocknum 4096//1time:2^22  18min:2^38  1day:2^44-2^45

__device__ int64_t hamming(int64_t n) {
    n = (n & 0x5555555555555555) + ((n >> 1) & 0x5555555555555555);
    n = (n & 0x3333333333333333) + ((n >> 2) & 0x3333333333333333);
    n = (n & 0x0f0f0f0f0f0f0f0f) + ((n >> 4) & 0x0f0f0f0f0f0f0f0f);
    n = (n & 0x00ff00ff00ff00ff) + ((n >> 8) & 0x00ff00ff00ff00ff);
    n = (n & 0x0000ffff0000ffff) + ((n >> 16) & 0x0000ffff0000ffff);
    n = (n & 0x00000000ffffffff) + ((n >> 32) & 0x00000000ffffffff);
    return n;
}

__global__ void simu(int64_t* a, int64_t* b, int* c, int t)//���ж������
{
    int offset;
    offset = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ int k;
    k= offset / 128 + t * MaxThreadPerBlock * Blocknum / 128;
    c[k * (M + 1) + offset%128] = hamming(a[offset%128] & b[k])%2;  
    atomicAdd(&c[k * (M + 1) + M], c[k * (M + 1) + offset%128]); 
}
__global__ void countweight(int64_t* b,int* c,int* result)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    if (c[k * (M + 1) + M] < 23)
    {
        atomicAdd(&result[c[k * (M + 1) + M]],1);
    }
    b[k] += MaxThreadPerBlock * Blocknum;
    //c[k * (M + 1) + M] = 0;
}

int main()
{
    hipError_t cudaStatus;
    hipDeviceProp_t prop;
    FILE* fp_test;
    int Num_Device;

    // ����ϵͳ�е�GPU����,��ָ��������һ��,ͬʱ�õ���GPU�����ܲ���
    cudaStatus = hipGetDeviceCount(&Num_Device);
    if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
    {
        printf("There is no GPU beyond 1.0, exit!\n");
        exit(0);
    }
    else
    {
        cudaStatus = hipGetDeviceProperties(&prop, Num_Device - 1);	// ѡ�����һ��GPU���ڼ���,ͬʱ����������ܲ���
        if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
        {
            printf("Cannot get device properties, exit!\n");
            exit(0);
        }
    }
    printf("Device Name : %s.\n", prop.name);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
    printf("maxThreadsPerMultiProcessor : %d.\n", prop.maxThreadsPerMultiProcessor);
    printf("Blocknum : %d.\n", Blocknum);
    int* h_AT = (int*)malloc(sizeof(int) * M * N);
    int64_t* h_A = (int64_t*)malloc(sizeof(int64_t) * M);
    int64_t* h_B = (int64_t*)malloc(sizeof(int64_t) * Blocknum * MaxThreadPerBlock);
    int* h_C = (int*)malloc(sizeof(int) * Blocknum * MaxThreadPerBlock * (M + 1));
    int* resultnum = (int*)malloc(sizeof(int) * 23);
    for (int i = 0; i < 23; i++)
    {
        resultnum[i] = 0;
    }

    fp_test = fopen("G_SPP.txt", "r");
    int h_num = 0;
    for (int i = 0; i < M * N; i++)
    {

        fscanf(fp_test, "%d", &h_num);//ÿM��Ԫ�أ�ֻ��¼һ����1��λ��        
        h_AT[i] = h_num;
    }
    fclose(fp_test);
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_A[i] = 2 * h_A[i] + h_AT[(63-j) * M + i];
        }
    }
    for (int i = 0; i < Blocknum * MaxThreadPerBlock; i++)
    {
        h_B[i] = -361976000113803264+i;   
    }

    int64_t* d_A, * d_B;
    int* d_C, * d_resultnum;
    hipMalloc((void**)&d_A, sizeof(int64_t) * M);
    hipMalloc((void**)&d_B, sizeof(int64_t) * Blocknum * MaxThreadPerBlock);
    hipMalloc((void**)&d_C, sizeof(int) * Blocknum * MaxThreadPerBlock * (M + 1));
    hipMalloc((void**)&d_resultnum, sizeof(int) * 23);

    hipMemcpy(d_A, h_A, M * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_resultnum, resultnum, 23 * sizeof(int), hipMemcpyHostToDevice);
    int64_t k = 0;
    clock_t start = clock();
    clock_t end;
    while (k < 1)//��
    {
        for (int i = 0; i < 128; i++)
        {
            simu << <Blocknum * MaxThreadPerBlock / 128, 128 >> > (d_A, d_B, d_C, i);
        }
        //hipDeviceSynchronize();
        countweight << < Blocknum, MaxThreadPerBlock >> > (d_B,d_C,d_resultnum);
        if (k%4096==0)
        {
            end = clock();
            printf("\ntime=%f min\n", (double)(end - start) / CLOCKS_PER_SEC/60);
           
            hipMemcpy(resultnum, d_resultnum, 23 * sizeof(int), hipMemcpyDeviceToHost);
            for (int i = 0; i < 23; i++)
            {
                cout << resultnum[i] << "  ";
            }
            cout << endl;
            cout << k << endl;
        }
        k++;
        //hipDeviceSynchronize();

    }
    end = clock();
    printf("\ntime=%f min\n", (double)(end - start) / CLOCKS_PER_SEC/60);

    hipMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, Blocknum * MaxThreadPerBlock * (M + 1) * sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<129;i++)
    {
        cout<<h_C[i]<<" ";
    }
    cout<<endl;
    cout << h_B[Blocknum * MaxThreadPerBlock - 1] << endl;
    hipMemcpy(resultnum, d_resultnum, 23 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 23; i++)
    {
        cout << resultnum[i] << "  ";
    }
    cout << endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_AT);
    free(h_B);
    free(h_C);
    return 0;
}