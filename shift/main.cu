#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
//#include <conio.h>
#include <time.h>
using namespace std;

#define M 128
#define N 64
#define MaxThreadPerBlock 1024
#define Blocknum 4096//1time:2^22  18min:2^38  1day:2^44-2^45

uint64_t checkrepeat(FILE *f,int name,int mode)
{
    uint64_t num;
    int k=0;
    int sum=0;
    uint64_t results[1000000];
    fseek(f,0,0);
    for(uint64_t i=0;!feof(f);i++)
    {
        k=0;
        fscanf(f, "%ld", &num);
        for(uint64_t t=0;t<sum;t++)
        {
            if(num==results[t])
            {
                k=1;
                    //cout<<num<<endl;
                break;
            }
        }
        if(k==0)
        {
            results[sum]=num;
            sum=sum+1;
            //cout<<sum<<endl;
        }
    }
    fclose(f);
    if(mode==1)
    {
        if(name==16)
        {
            f=fopen("16_1.txt","w+");
        }
        else if(name==18)
        {
            f=fopen("18_1.txt","w+");
        }
        else if(name==20)
        {
            f=fopen("20_1.txt","w+");
        }
        else if(name==22)
        {
            f=fopen("22_1.txt","w+");
        }
        for(uint64_t i=0;i<sum;i++)
        {
            fprintf(f,"%ld ",results[i]);
        }
        fprintf(f,"\n\n");
        fclose(f);
        printf("saved\n");
    }
    if(name==16)
    {
        f=fopen("16.txt","w+");
    }
    else if(name==18)
    {
        f=fopen("18.txt","w+");
    }
    else if(name==20)
    {
        f=fopen("20.txt","w+");
    }
    else if(name==22)
    {
        f=fopen("22.txt","w+");
    }
    for(uint64_t i=0;i<sum;i++)
    {
        fprintf(f,"%ld ",results[i]);
    }
    fprintf(f,"\n\n");
    //fclose(f);
    return sum;
}

__device__ uint64_t shiftr(uint64_t n,int t) {
    n =(n<<(64-t))|(n>>t);
    return n;
}
__device__ uint64_t shiftl(uint64_t n,int t) {
    n =(n>>(64-t))|(n<<t);
    return n;
}

__device__ uint64_t hamming(uint64_t n) {
    n = (n & 0x5555555555555555) + ((n >> 1) & 0x5555555555555555);
    n = (n & 0x3333333333333333) + ((n >> 2) & 0x3333333333333333);
    n = (n & 0x0f0f0f0f0f0f0f0f) + ((n >> 4) & 0x0f0f0f0f0f0f0f0f);
    n = (n & 0x00ff00ff00ff00ff) + ((n >> 8) & 0x00ff00ff00ff00ff);
    n = (n & 0x0000ffff0000ffff) + ((n >> 16) & 0x0000ffff0000ffff);
    n = (n & 0x00000000ffffffff) + ((n >> 32) & 0x00000000ffffffff);
    return n;
}

__global__ void simu(uint64_t* a, uint64_t* b, int* c,int t,int* weight)//���ж������
{
    int offset;
    offset = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ int k;
    k= offset / 128 + t * MaxThreadPerBlock * Blocknum / 128;
    //c[k * M + offset%128] = hamming(a[offset%128] & b[k])%2;  
    atomicAdd(&weight[k], hamming(a[offset%128] & b[k])%2);
}
__global__ void gpushiftl(uint64_t* c,int t)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    c[k]=shiftl(c[k],t);
}
__global__ void gpushiftr(uint64_t* c,int t)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    c[k]=shiftr(c[k],t);
}
__global__ void countweight(int* c,int* result)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    if (c[k] < 23)
    {
        atomicAdd(&result[c[k]],1);
    }

}
__global__ void hbadd(uint64_t* b)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    b[k] += MaxThreadPerBlock * Blocknum;
}
__global__ void clearzero(int* c)
{
    int k = 0;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    c[k] = 0;
}
int main()
{
    srand(time(NULL));
    hipError_t cudaStatus;
    hipDeviceProp_t prop;
    FILE* fp_test;
    int Num_Device;

    // ����ϵͳ�е�GPU����,��ָ��������һ��,ͬʱ�õ���GPU�����ܲ���
    cudaStatus = hipGetDeviceCount(&Num_Device);
    if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
    {
        printf("There is no GPU beyond 1.0, exit!\n");
        exit(0);
    }
    else
    {
        cudaStatus = hipGetDeviceProperties(&prop, Num_Device - 1);	// ѡ�����һ��GPU���ڼ���,ͬʱ����������ܲ���
        if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
        {
            printf("Cannot get device properties, exit!\n");
            exit(0);
        }
    }
    printf("Device Name : %s.\n", prop.name);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
    printf("maxThreadsPerMultiProcessor : %d.\n", prop.maxThreadsPerMultiProcessor);
    printf("Blocknum : %d.\n", Blocknum);
    int* h_AT = (int*)malloc(sizeof(int) * M * N);
    uint64_t* h_A = (uint64_t*)malloc(sizeof(uint64_t) * M);
    uint64_t* h_A1 = (uint64_t*)malloc(sizeof(uint64_t) * M);
    uint64_t* h_B = (uint64_t*)malloc(sizeof(uint64_t) * Blocknum * MaxThreadPerBlock);
    int* h_C = (int*)malloc(sizeof(int) * Blocknum * MaxThreadPerBlock * M);
    int* weight = (int*)malloc(sizeof(int) * Blocknum * MaxThreadPerBlock);
    int* resultnum = (int*)malloc(sizeof(int) * 23);
   
    for (int i = 0; i < 23; i++)
    {
        resultnum[i] = 0;
    }

    fp_test = fopen("G_PAC.txt", "r");
    int h_num = 0;
    for (int i = 0; i < M * N; i++)
    {

        fscanf(fp_test, "%d", &h_num);//ÿM��Ԫ�أ�ֻ��¼һ����1��λ��        
        h_AT[i] = h_num;
    }
    fclose(fp_test);
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_A[i] = 2 * h_A[i] + h_AT[(63-j) * M + i];
            h_A1[i] = 2 * h_A1[i] + h_AT[j* M + i];
        }
    }
    for (int i = 0; i < Blocknum * MaxThreadPerBlock; i++)
    {
        h_B[i] = (uint64_t)19406846/2*4096*1024+1+i;   
    }

    uint64_t* d_A, * d_B,* d_A1;
    int* d_C, * d_resultnum,* d_weight;
    hipMalloc((void**)&d_weight, sizeof(int) * Blocknum * MaxThreadPerBlock);
    hipMalloc((void**)&d_A, sizeof(uint64_t) * M);
    hipMalloc((void**)&d_A1, sizeof(uint64_t) * M);
    hipMalloc((void**)&d_B, sizeof(uint64_t) * Blocknum * MaxThreadPerBlock);
    hipMalloc((void**)&d_C, sizeof(int) * Blocknum * MaxThreadPerBlock * M);
    hipMalloc((void**)&d_resultnum, sizeof(int) * 23);
   


    hipMemcpy(d_A, h_A, M * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_A1, h_A1, M * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Blocknum * MaxThreadPerBlock * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_resultnum, resultnum, 23 * sizeof(int), hipMemcpyHostToDevice);
    uint64_t k = 19406846;
    uint64_t k1=k+4096;
    clock_t start = clock();
    clock_t end;
    FILE* fp_16;
    FILE* fp_18;
    FILE* fp_20;
    FILE* fp_22;
    fp_16=fopen("16.txt","a+");
    fp_18=fopen("18.txt","a+");
    fp_20=fopen("20.txt","a+");
    fp_22=fopen("22.txt","a+");
    fp_test=fopen("results.txt","a+");
    int shiftnum=0;
    while (k<k1)//��
    //while(k<256)
    {
            for (int i = 0; i < 128; i++)
            {
                simu << <Blocknum * MaxThreadPerBlock / 128, 128 >> > (d_A, d_B,d_C,i,d_weight);
            }
        
        //hipDeviceSynchronize();
        countweight << < Blocknum, MaxThreadPerBlock >> > (d_weight,d_resultnum);
//       hipMemcpy(h_C, d_C, Blocknum * MaxThreadPerBlock * (M + 1) * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(weight, d_weight, Blocknum * MaxThreadPerBlock * sizeof(int), hipMemcpyDeviceToHost);
       hipMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(int), hipMemcpyDeviceToHost);
            for(int i=0;i < MaxThreadPerBlock * Blocknum;i++)
            {
            
                    if(weight[i]==16)
                    {
                        fprintf(fp_16,"%ld ",h_B[i]);
                    }
                    else if(weight[i]==18)
                    {
                        fprintf(fp_18,"%ld ",h_B[i]);
                    }
                    else if(weight[i]==20)
                    {
                        fprintf(fp_20,"%ld ",h_B[i]);
                    }
                    else if(weight[i]==22)
                    {
                        fprintf(fp_22,"%ld ",h_B[i]);
                    }
            }

        if(k%2==0)
        {
            shiftnum=rand()%21+10;
            gpushiftl << < Blocknum, MaxThreadPerBlock >> > (d_B,shiftnum);
            //hbadd << < Blocknum, MaxThreadPerBlock >> > (d_B);
        }
        else
        {
            gpushiftr << < Blocknum, MaxThreadPerBlock >> > (d_B,shiftnum);
            //hipDeviceSynchronize();
            hbadd << < Blocknum, MaxThreadPerBlock >> > (d_B);
        }
        clearzero << < Blocknum, MaxThreadPerBlock >> > (d_weight);
        if ((k+1)%1024==0)
        {
            if ((k+1)%4096)
            {
                checkrepeat(fp_16,16,0);
                checkrepeat(fp_18,18,0);
                checkrepeat(fp_20,20,0);
                checkrepeat(fp_22,22,0);
                hipMemcpy(resultnum, d_resultnum, 23 * sizeof(int), hipMemcpyDeviceToHost);
            }
            else
            {
                srand(time(NULL));
                resultnum[16]=checkrepeat(fp_16,16,1);
                resultnum[18]=checkrepeat(fp_18,18,1);
                resultnum[20]=checkrepeat(fp_20,20,1);
                resultnum[22]=checkrepeat(fp_22,22,1);
            }
            end = clock();
            printf("\ntime=%f min\n", (double)(end - start) / CLOCKS_PER_SEC/60);
           
            //hipMemcpy(resultnum, d_resultnum, 23 * sizeof(int), hipMemcpyDeviceToHost);
            fprintf(fp_test,"k = %ld\n",k);
            for (int i = 0; i < 23; i++)
            {
                cout << resultnum[i] << "  ";
                fprintf(fp_test,"%d ",resultnum[i]);
            }
            cout << endl;
            hipMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(uint64_t), hipMemcpyDeviceToHost);
            printf("last b: %ld\n",h_B[Blocknum * MaxThreadPerBlock - 1]);
            fprintf(fp_test,"\n");
            fprintf(fp_test,"last b: %ld\n",h_B[Blocknum * MaxThreadPerBlock - 1]);
            printf("k: %ld\n",k);            
            fclose(fp_test);
            fp_test=fopen("results.txt","a");
        }
        k++;
        //hipDeviceSynchronize();
        //cout<<k<<endl;
    }
    end = clock();
    printf("\ntime=%f min\n", (double)(end - start) / CLOCKS_PER_SEC/60);
    hipMemcpy(h_B, d_B, Blocknum * MaxThreadPerBlock * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, Blocknum * MaxThreadPerBlock * M * sizeof(int), hipMemcpyDeviceToHost);
    printf("last b: %ld\n",h_B[Blocknum * MaxThreadPerBlock - 1]);
    //hipMemcpy(resultnum, d_resultnum, 23 * sizeof(int), hipMemcpyDeviceToHost);
    cout<<endl;
    // for(int i=0;i<10;i++)
    // {
    //     cout<<h_B[i]<<" ";
    // }
    // cout<<endl;
    // for(int i=0;i<2000;i++)
    // {
    //     cout<<h_C[i]<<" ";
    //     if((i+1)%128==0)
    //     {
    //         cout<<endl<<endl;
    //     }
    // }
    fprintf(fp_test,"k = %ld\n",k);
    for (int i = 0; i < 23; i++)
    {
        cout << resultnum[i] << "  ";
        fprintf(fp_test,"%d ",resultnum[i]);
    }
    fprintf(fp_test,"\n");
    cout << endl;
    fclose(fp_16);
    fclose(fp_18);
    fclose(fp_20);
    fclose(fp_22);
    fclose(fp_test);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_AT);
    free(h_B);
    free(h_C);
    return 0;
}